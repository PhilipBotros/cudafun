extern "C" {
#include <hip/hip_runtime.h>

__global__ void add_arrays(const float* a, const float* b, float* out, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
        out[idx] = a[idx] + b[idx];
}

void launch_sum_kernel(const float* a, const float* b, float* out, int n) {
    float *d_a, *d_b, *d_out;
    size_t size = n * sizeof(float);

    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_out, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (n + threads - 1) / threads;
    add_arrays<<<blocks, threads>>>(d_a, d_b, d_out, n);

    hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
}
}