#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <cmath>

// Kernel code
// A is input matrix, K is conv kernel, C is output matrix 
// Assume A=MxN and B=KxK -> C=MxN
__global__ void basicConv(float *d_A, float *d_K, float *d_C, int M, int N, int K) {
    int col = blockIdx.x * blockDim.x + threadIdx.x; 
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Skip threads outside the output matrix bounds
    if (row >= M || col >= N) {
        return;
    }

    int start_idx = -K / 2;   // Start of kernel offset
    int end_idx = K / 2;      // End of kernel offset

    float acc = 0.0f;
    for (int i = start_idx; i <= end_idx; i++) {
        for (int j = start_idx; j <= end_idx; j++) {
            int A_row = row + i;   // Input matrix row index
            int A_col = col + j;   // Input matrix column index

            // Check input matrix bounds
            if (A_row >= 0 && A_row < M && A_col >= 0 && A_col < N) {
                acc += d_A[A_row * N + A_col] * d_K[(i + end_idx) * K + (j + end_idx)];
            }
        }
    }
    d_C[row * N + col] = acc;
}

// CPU conv for verification
void cpuConv(float *A, float *K, float *C, int M, int N, int kernel_size) {
    int start_idx = -kernel_size / 2;
    int end_idx = kernel_size / 2;

    for (int row = 0; row < M; row++) {
        for (int col = 0; col < N; col++) {
            float acc = 0.0f;
            for (int i = start_idx; i <= end_idx; i++) {
                for (int j = start_idx; j <= end_idx; j++) {
                    int A_row = row + i; 
                    int A_col = col + j; 

                    if (A_row >= 0 && A_row < M && A_col >= 0 && A_col < N) {
                        acc += A[A_row * N + A_col] * K[(i + end_idx) * kernel_size + (j + end_idx)];
                    }
                }
            }
            C[row * N + col] = acc;
        }
    }
}

// Verification function
bool verifyResults(float *gpu_result, float *cpu_result, int M, int N, float tolerance = 1e-5) {
    for (int i = 0; i < M * N; i++) {
        if (std::fabs(gpu_result[i] - cpu_result[i]) > tolerance) {
            std::cout << "Mismatch at position " << i << ": GPU = " << gpu_result[i] 
                      << ", CPU = " << cpu_result[i] << std::endl;
            return false;
        }
    }
    return true;
}

int main() {
    // Create data on host
    // Create matrix and conv kernel and fill with random floats [0.0, 1.0)
    const int M = 100, N = 100, K = 5; 
    float *h_A = new float[M * N];
    float *h_K = new float[K * K];
    float *h_C = new float[M * N];
    float *cpu_C = new float[M * N];  // For CPU verification

    std::mt19937 gen(42);
    std::uniform_real_distribution<float> dis(0.0, 1.0); 
    
    for (int i = 0; i < M * N; ++i) {
        h_A[i] = dis(gen);
    }
    for (int i = 0; i < K * K; ++i) {
        h_K[i] = dis(gen);
    }

    // Calculate sizes for memory allocation
    size_t bytes_A = M * N * sizeof(float);
    size_t bytes_K = K * K * sizeof(float);
    size_t bytes_C = M * N * sizeof(float);

    // Allocate device memory
    float *d_A, *d_K, *d_C;
    hipMalloc((void **)&d_A, bytes_A);
    hipMalloc((void **)&d_K, bytes_K);
    hipMalloc((void **)&d_C, bytes_C);

    // Copy matrices from host to device
    hipMemcpy(d_A, h_A, bytes_A, hipMemcpyHostToDevice);
    hipMemcpy(d_K, h_K, bytes_K, hipMemcpyHostToDevice);
    
    // Launch kernel
    const int blockDim = 16;
    dim3 blockSize(blockDim, blockDim);
    dim3 gridSize((N + blockDim - 1) / blockDim, (M + blockDim - 1) / blockDim);
    basicConv<<<gridSize, blockSize>>>(d_A, d_K, d_C, M, N, K);

    // Return result from device to host
    hipMemcpy(h_C, d_C, bytes_C, hipMemcpyDeviceToHost);

    // Compute CPU result
    cpuConv(h_A, h_K, cpu_C, M, N, K);

    // Verify results
    bool passed = verifyResults(h_C, cpu_C, M, N);
    if (passed) {
        std::cout << "Verification PASSED!" << std::endl;
    } else {
        std::cout << "Verification FAILED!" << std::endl;
    }

    // Clean up
    hipFree(d_A);
    hipFree(d_K);
    hipFree(d_C);
    delete[] h_A;
    delete[] h_K;
    delete[] h_C;
    delete[] cpu_C;
    
    return 0;
}